#include <stdio.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <cassert>
#include <cstdlib>
#include <functional>
#include <iostream>
#include <algorithm>
#include <vector>

using std::cout;
using std::generate;
using std::vector;

#define SIZE 1000
#define N 10
#define RANDMAX 65536

#define CUDA_CALL(x) do { if((x)!=hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    return EXIT_FAILURE;}} while(0)

#define CHECK(x) do { if((x)!=hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    return EXIT_FAILURE;}} while(0)

__global__ void computeMovingAverage(float *dev_a, float *dev_b, int size, int n) {

	int idx = blockDim.x * blockIdx.x + threadIdx.x; // 0, 1, 2 
	int halfway = size/2; // 1000/2
	int i = halfway; 
	
	do{
		if(idx < i)
  			if(dev_a[idx+i] > dev_a[idx])
  				dev_a[idx] = dev_a[idx+i]; 
  		 
  		__syncthreads(); 

  		if(i%2==0) 
  			i = i/2; 
  		else 
  			i = (i+1)/2; 

	} while (i > 1);
	
	__syncthreads(); 

	if(dev_a[1]>dev_a[0])
		dev_a[0] = dev_a[1];
	
}

void computeMovingAverageOnCPU(vector<float> &host_a, float &cpuRef, const int size) {	

	int maximum = host_a[0]; 

	for(int i = 0; i < size; i++){

		if(host_a[i] >= maximum){
			maximum = host_a[i]; 		
		}

	}
	cpuRef = maximum; 
	
}

int main(void){

	// set up device
	int dev = 0; 
	hipDeviceProp_t deviceProp; 
	CHECK(hipGetDeviceProperties(&deviceProp, dev));
	printf("Using Device %d: %s\n", dev, deviceProp.name);
	CHECK(hipSetDevice(dev)); 

	int n = N; 
	int size = SIZE; 
	int randmax = RANDMAX;

	printf("Array Size: %d  Sample Size: %d\n", size, N);
	size_t nBytes = size * sizeof(float); 
	float cpuRef = 0.0f; 

	// initialize random number
	srand ((int)time(0));
 
	// initialize vector and generate random indices between 0 and 5. 
	vector<float> host_a(size);
	vector<float> host_b(size); 
	printf("Generating %d random integers from 0 to %d\n", size, randmax); 
	generate(host_a.begin(), host_a.end(), []() { return rand() % RANDMAX; }); 

	float *dev_a, *dev_b; 
	hipMalloc(&dev_a, nBytes); 
	hipMalloc(&dev_b, nBytes); 
	hipMemcpy(dev_a, host_a.data(), nBytes, hipMemcpyHostToDevice); 
	hipMemcpy(dev_b, host_b.data(), nBytes, hipMemcpyHostToDevice); 
	// declare block and grid dimension. 

	dim3 block (size/n); 
	dim3 grid (n); 

	// Timer starts 
	float GPUtime, CPUtime; 
	hipEvent_t start, stop; 

	hipEventCreate(&start); 
	hipEventCreate(&stop); 
	hipEventRecord(start, 0); 

	printf("Launching Kernel \n"); 
	computeMovingAverage <<< grid, block >>> (dev_a, dev_b, size, n); 
	
	hipMemcpy(host_a.data(), dev_a, nBytes, hipMemcpyDeviceToHost); 

	// timer stops
	hipEventRecord(stop, 0); 
	hipEventSynchronize(stop); 
	hipEventElapsedTime(&GPUtime, start, stop); 

	hipEventCreate(&start); 
	hipEventCreate(&stop); 
	hipEventRecord(start, 0); 

	computeMovingAverageOnCPU(host_a, cpuRef, size);

	hipEventRecord(stop, 0); 
	hipEventSynchronize(stop); 
	hipEventElapsedTime(&CPUtime, start, stop); 

    printf("Kernel: computeMovingAverage <<<gridDim: %d, blockDim: %d>>>\n", grid.x, block.x); 

	printf("Compute time on GPU: %3.6f ms \n", GPUtime); 
	printf("Compute time on CPU: %3.6f ms \n", CPUtime); 
	printf("Maximum integer found on CPU: %d\n", (int)cpuRef); 
	printf("Maximum integer found on GPU: %d\n", (int)host_a[0]); 

	hipFree(dev_a);
	hipFree(dev_b); 

	return (0); 
}